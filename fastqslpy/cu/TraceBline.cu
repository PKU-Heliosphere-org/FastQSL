#include "hip/hip_runtime.h"
/**
*    @file TraceBline.cu
*    @author Peijin Zhang
*    The kernel of the Q-factor computation: tracing the magnetic field
*/
#include <math.h>
#include <stdio.h>
#include "hip/hip_vector_types.h"  /// Math API
//#include "TraceBline.cuh"
#define M_PI 3.14159265   ///< Mathematical constant PI.
#define MAX_STEP_RATIO 4  ///< Maximum step length compared to box size.
#define MAX_NUM_STEP 500000 ///< Maximum number of steps.

extern "C"{
inline __device__ float lenVec3xyz(float xx,float yy,float zz){
    return sqrtf(xx*xx + yy*yy + zz*zz);}

inline __device__ float lenVec3(float3 a){
    return sqrtf(a.x*a.x + a.y*a.y + a.z*a.z);}

inline __device__ float dot3(float3 a, float3 b)
{  return a.x*b.x + a.y*b.y + a.z*b.z;}

inline __device__ double ddot3(float3 a, float3 b)
{  return (double)a.x*b.x + (double)a.y*b.y + (double)a.z*b.z;}

inline __device__ float3 divide3(float3 a, float b)
{  return make_float3(a.x/b , a.y/b , a.z/b);}

inline __device__ float3 normalize3(float4 a)
{  return make_float3(a.x/a.w,  a.y/a.w,  a.z/a.w);}

__forceinline__ __device__ float get_Idx3d(float *Arr,int3 AShapeN3,int Idx0,int Idx1,int Idx2){
    //return Arr[Idx0* AShapeN[1]*AShapeN[2]  +  Idx1* AShapeN[2]  +  Idx2];
    return Arr[Idx2* AShapeN3.y*AShapeN3.x  +  Idx1*AShapeN3.x  +  Idx0];
}

__global__ void  test_Idx3d(float *Arr,int *AShapeN, int *getIdx,float *res){
    printf("idx : %d    %d    %d   ", getIdx[0],getIdx[1],getIdx[2]);
    int3 AShapeN3 = make_int3(AShapeN[0],AShapeN[1],AShapeN[2]);
    res[0] = get_Idx3d(Arr,AShapeN3,getIdx[0],getIdx[1],getIdx[2]);
}

__device__ float Interp3d(float *Arr,int3 AShapeN3, \
    float inPoint_0, float inPoint_1, float inPoint_2){
    //algorithm [https://core.ac.uk/download/pdf/44386053.pdf]
    float rx,ry,rz; // ratio of the point
    float Arr000,Arr001,Arr010,Arr011,Arr100,Arr101,Arr110,Arr111;
    float Aget;
    int x_Idx,y_Idx,z_Idx;

    // handle out of boundary problem by extending
    inPoint_0 = (inPoint_0>0) ? inPoint_0 : 0.0001;
    inPoint_1 = (inPoint_1>0) ? inPoint_1 : 0.0001;
    inPoint_2 = (inPoint_2>0) ? inPoint_2 : 0.0001;
    inPoint_0 = (inPoint_0<AShapeN3.x-1) ? inPoint_0 : ((float)AShapeN3.x-1-0.0001);
    inPoint_1 = (inPoint_1<AShapeN3.y-1) ? inPoint_1 : ((float)AShapeN3.y-1-0.0001);
    inPoint_2 = (inPoint_2<AShapeN3.z-1) ? inPoint_2 : ((float)AShapeN3.z-1-0.0001);

    // ratio of the points to adjacent grid
    rx = inPoint_0-floorf(inPoint_0);
    ry = inPoint_1-floorf(inPoint_1);
    rz = inPoint_2-floorf(inPoint_2);

    // index of point in the down-side
    x_Idx = __float2int_rd(inPoint_0);
    y_Idx = __float2int_rd(inPoint_1);
    z_Idx = __float2int_rd(inPoint_2);
    
    //printf("x,y,z %f %f %f\n",inPoint_0,inPoint_1,inPoint_2);
    //printf("x,y,z %d %d %d\n",x_Idx,y_Idx,z_Idx);

    // grid boundary
    Arr000 = get_Idx3d(Arr,AShapeN3, x_Idx  ,y_Idx  ,z_Idx  );
    Arr001 = get_Idx3d(Arr,AShapeN3, x_Idx  ,y_Idx  ,z_Idx+1);
    Arr010 = get_Idx3d(Arr,AShapeN3, x_Idx  ,y_Idx+1,z_Idx  );
    Arr011 = get_Idx3d(Arr,AShapeN3, x_Idx  ,y_Idx+1,z_Idx+1);
    Arr100 = get_Idx3d(Arr,AShapeN3, x_Idx+1,y_Idx  ,z_Idx  );
    Arr101 = get_Idx3d(Arr,AShapeN3, x_Idx+1,y_Idx  ,z_Idx+1);
    Arr110 = get_Idx3d(Arr,AShapeN3, x_Idx+1,y_Idx+1,z_Idx  );
    Arr111 = get_Idx3d(Arr,AShapeN3, x_Idx+1,y_Idx+1,z_Idx+1);

    Aget =      Arr000* (1.-rx)*(1.-ry)*(1.-rz)+\
                Arr001* (1.-rx)*(1.-ry)*(  rz)+\
                Arr010* (1.-rx)*(  ry)*(1.-rz)+\
                Arr011* (1.-rx)*(  ry)*(  rz)+\
                Arr100* (  rx)*(1.-ry)*(1.-rz)+\
                Arr101* (  rx)*(1.-ry)*(  rz)+\
                Arr110* (  rx)*(  ry)*(1.-rz)+\
                Arr111* (  rx)*(  ry)*(  rz);
    return Aget;
}

inline __device__ float3 Interp3dxyzn(float *Arr_x,float *Arr_y,float *Arr_z,\
    int3 AShapeN3, float3 inPoint_this,bool norm_flag){
    // normalized B interpolation
    //algorithm [https://core.ac.uk/download/pdf/44386053.pdf]
    float rx,ry,rz; // ratio of the point
    float Arr000_x,Arr001_x,Arr010_x,Arr011_x,Arr100_x,Arr101_x,Arr110_x,Arr111_x;
    float Arr000_y,Arr001_y,Arr010_y,Arr011_y,Arr100_y,Arr101_y,Arr110_y,Arr111_y;
    float Arr000_z,Arr001_z,Arr010_z,Arr011_z,Arr100_z,Arr101_z,Arr110_z,Arr111_z;
    float w000,  w001,  w010,  w011,  w100,  w101,  w110,  w111;
    int x_Idx,y_Idx,z_Idx;
    float norm_B;
    float3 res;

    float inPoint_0=inPoint_this.x;
    float inPoint_1=inPoint_this.y;
    float inPoint_2=inPoint_this.z;
    // ratio of the points to adjacent grid
    rx = inPoint_0-floorf(inPoint_0);
    ry = inPoint_1-floorf(inPoint_1);
    rz = inPoint_2-floorf(inPoint_2);
    // index of point in the down-side
    x_Idx = __float2int_rd(inPoint_0);
    y_Idx = __float2int_rd(inPoint_1);
    z_Idx = __float2int_rd(inPoint_2);
    
    if (inPoint_0<= 0.){
        rx=0; x_Idx=0;}
    if (inPoint_0>= AShapeN3.x -1){
        rx=1; x_Idx=AShapeN3.x -2;}
    if (inPoint_1<= 0.){
        ry=0; y_Idx=0;}
    if (inPoint_1>= AShapeN3.y -1){
        ry=1; y_Idx=AShapeN3.y -2;}
    if (inPoint_2<= 0.){
        rz=0.; z_Idx=0;}
    if (inPoint_2>= AShapeN3.z -1){
        rz=1.; z_Idx=AShapeN3.z -2;}
    
    // calculate the weight of the point and use for three times
    w000 = (1.-rx)*(1.-ry)*(1.-rz);
    w001 = (1.-rx)*(1.-ry)*(   rz);
    w010 = (1.-rx)*(   ry)*(1.-rz);
    w011 = (1.-rx)*(   ry)*(   rz);
    w100 = (   rx)*(1.-ry)*(1.-rz);
    w101 = (   rx)*(1.-ry)*(   rz);
    w110 = (   rx)*(   ry)*(1.-rz);
    w111 = (   rx)*(   ry)*(   rz);

    // grid boundary
    Arr000_x = get_Idx3d(Arr_x,AShapeN3, x_Idx  ,y_Idx  ,z_Idx  );
    Arr001_x = get_Idx3d(Arr_x,AShapeN3, x_Idx  ,y_Idx  ,z_Idx+1);
    Arr010_x = get_Idx3d(Arr_x,AShapeN3, x_Idx  ,y_Idx+1,z_Idx  );
    Arr011_x = get_Idx3d(Arr_x,AShapeN3, x_Idx  ,y_Idx+1,z_Idx+1);
    Arr100_x = get_Idx3d(Arr_x,AShapeN3, x_Idx+1,y_Idx  ,z_Idx  );
    Arr101_x = get_Idx3d(Arr_x,AShapeN3, x_Idx+1,y_Idx  ,z_Idx+1);
    Arr110_x = get_Idx3d(Arr_x,AShapeN3, x_Idx+1,y_Idx+1,z_Idx  );
    Arr111_x = get_Idx3d(Arr_x,AShapeN3, x_Idx+1,y_Idx+1,z_Idx+1);
    
    Arr000_y = get_Idx3d(Arr_y,AShapeN3, x_Idx  ,y_Idx  ,z_Idx  );
    Arr001_y = get_Idx3d(Arr_y,AShapeN3, x_Idx  ,y_Idx  ,z_Idx+1);
    Arr010_y = get_Idx3d(Arr_y,AShapeN3, x_Idx  ,y_Idx+1,z_Idx  );
    Arr011_y = get_Idx3d(Arr_y,AShapeN3, x_Idx  ,y_Idx+1,z_Idx+1);
    Arr100_y = get_Idx3d(Arr_y,AShapeN3, x_Idx+1,y_Idx  ,z_Idx  );
    Arr101_y = get_Idx3d(Arr_y,AShapeN3, x_Idx+1,y_Idx  ,z_Idx+1);
    Arr110_y = get_Idx3d(Arr_y,AShapeN3, x_Idx+1,y_Idx+1,z_Idx  );
    Arr111_y = get_Idx3d(Arr_y,AShapeN3, x_Idx+1,y_Idx+1,z_Idx+1);
    
    Arr000_z = get_Idx3d(Arr_z,AShapeN3, x_Idx  ,y_Idx  ,z_Idx  );
    Arr001_z = get_Idx3d(Arr_z,AShapeN3, x_Idx  ,y_Idx  ,z_Idx+1);
    Arr010_z = get_Idx3d(Arr_z,AShapeN3, x_Idx  ,y_Idx+1,z_Idx  );
    Arr011_z = get_Idx3d(Arr_z,AShapeN3, x_Idx  ,y_Idx+1,z_Idx+1);
    Arr100_z = get_Idx3d(Arr_z,AShapeN3, x_Idx+1,y_Idx  ,z_Idx  );
    Arr101_z = get_Idx3d(Arr_z,AShapeN3, x_Idx+1,y_Idx  ,z_Idx+1);
    Arr110_z = get_Idx3d(Arr_z,AShapeN3, x_Idx+1,y_Idx+1,z_Idx  );
    Arr111_z = get_Idx3d(Arr_z,AShapeN3, x_Idx+1,y_Idx+1,z_Idx+1);

    res.x=Arr000_x* w000+\
            Arr001_x* w001+\
            Arr010_x* w010+\
            Arr011_x* w011+\
            Arr100_x* w100+\
            Arr101_x* w101+\
            Arr110_x* w110+\
            Arr111_x* w111;
    res.y=Arr000_y* w000+\
            Arr001_y* w001+\
            Arr010_y* w010+\
            Arr011_y* w011+\
            Arr100_y* w100+\
            Arr101_y* w101+\
            Arr110_y* w110+\
            Arr111_y* w111;
    res.z=Arr000_z* w000+\
            Arr001_z* w001+\
            Arr010_z* w010+\
            Arr011_z* w011+\
            Arr100_z* w100+\
            Arr101_z* w101+\
            Arr110_z* w110+\
            Arr111_z* w111;
    if(norm_flag){
        norm_B = sqrtf(res.x*res.x+res.y*res.y+res.z*res.z);
        res.x = res.x/norm_B;
        res.y = res.y/norm_B;
        res.z = res.z/norm_B;}
    return res;
}


__global__ void test_Interp3dxyz(float *Arr_x,float *Arr_y,float *Arr_z,int *AShapeN, \
    float *inPoint_0, float *inPoint_1, float *inPoint_2){
        float3 res,pointthis;
        pointthis=make_float3(inPoint_0[0],inPoint_1[0],inPoint_2[0]);
        int3 shapeshape = make_int3(AShapeN[0],AShapeN[1],AShapeN[2]);
        res = Interp3dxyzn(Arr_x,Arr_y,Arr_z,shapeshape,pointthis,true);
    }


__device__ void stepForward(float *Bx,float *By,float *Bz,int3 BshapeN3,\
    float *P_start, float *P_end,float s_len){
    float Bx_cur,By_cur,Bz_cur,B0;
    float3 Btmp;
    Bx_cur  = Interp3d(Bx,BshapeN3,P_start[0],P_start[1],P_start[2]);
    By_cur  = Interp3d(By,BshapeN3,P_start[0],P_start[1],P_start[2]);
    Bz_cur  = Interp3d(Bz,BshapeN3,P_start[0],P_start[1],P_start[2]);
    Btmp = make_float3(Bx_cur,By_cur,Bz_cur);
    B0 = lenVec3(Btmp);
    P_end[0] = P_start[0] + s_len*Bx_cur/B0;
    P_end[1] = P_start[1] + s_len*By_cur/B0;
    P_end[2] = P_start[2] + s_len*Bz_cur/B0;
    //printf("Inr:%f  :%f  :%f\n",s_len*Bx_cur/B0,s_len*By_cur/B0,s_len*Bz_cur/B0);
}
inline __device__ float3 RK4(float *Bx,float *By,float *Bz,int3 BshapeN3, float3 P0, float s_len){
    float3 k1,k2,k3,k4,P_end;
    k1 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0,true);
    k2 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k1/2.,true);
    k3 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k2/2.,true);
    k4 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k3,true);
    P_end = P0+1./6.*s_len*(k1 +2.*k2 +2.*k3 +k4);
    return P_end;
}

inline __device__ float selectFloat3xyz(float3 a, int dim){
    float res;
    switch(dim) {
        case 0  : res = a.x;  break;
        case 1  : res = a.y;  break;
        case 2  : res = a.z;  break;}
    return res;
}
inline __device__ float selectInt3xyz(int3 a, int dim){
    int res;
    switch(dim) {
        case 0  : res = a.x;  break;
        case 1  : res = a.y;  break;
        case 2  : res = a.z;  break;}
    return res;
}

inline __device__ float3 RK4_boundary(float *Bx,float *By,float *Bz,int3 BshapeN3, float3 P0, float s_len,int b_dim){
    float B0_k1,B0_k2,B0_k3,B0_k4;
    float3 Bk1,Bk2,Bk3,Bk4,P_end,k1,k2,k3,k4;
    // k1
    Bk1 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0,true);
    B0_k1 = selectFloat3xyz(Bk1,b_dim);
    k1 = Bk1/B0_k1;
    // k2
    Bk2 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k1/2.,true);
    B0_k2 = selectFloat3xyz(Bk2,b_dim);
    k2 = Bk2/B0_k2;
    // k3
    Bk3  = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k2/2.,true);
    B0_k3 = selectFloat3xyz(Bk3,b_dim);
    k3 = Bk3/B0_k3;
    // k4
    Bk4 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k3,true);
    B0_k4 = selectFloat3xyz(Bk4,b_dim);
    k4 = Bk4/B0_k4;
    P_end = P0 + (1./6.)* s_len*( k1 + 2.0*k2 + 2.0*k3 + k4);
    return P_end;
}

inline __device__ int checkFlag(int3 BshapeN3, float3 P_cur){
    // check current status
    int flag_res = 42; // 42 means un-categorized
    // flag=0 means inside running box
    if (P_cur.x>=0. &P_cur.y>0. &P_cur.z>=0. &  \
        P_cur.x<=BshapeN3.x-1. &P_cur.y<=BshapeN3.y-1. & P_cur.z<=BshapeN3.z-1. ){flag_res=0;} 
    else{ // ouside
        if (P_cur.x< 0.             &P_cur.y>=0.            &P_cur.z>=0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=1;} // x min 
        if (P_cur.x>=0.             &P_cur.y>=0.            &P_cur.z>=0. &  \
            P_cur.x>=BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=2;} // x max
        if (P_cur.x>=0.             &P_cur.y< 0.            &P_cur.z>=0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=3;} // y min
        if (P_cur.x>=0.             &P_cur.y>=0.            &P_cur.z>=0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y>=BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=4;} // y max 
        if (P_cur.x>=0.             &P_cur.y>=0.            &P_cur.z< 0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=5;} // z min
        if (P_cur.x>=0.             &P_cur.y>=0.            &P_cur.z>=0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z>=BshapeN3.z-1.   ){flag_res=6;} // z max
    }
    // dim = int((flag-1)/2)
    return flag_res;
}

__device__ void TraceBline(float *Bx,float *By,float *Bz,int3 BshapeN3,\
    float *curB_x, float *curB_y,  float *curB_z,double *twist_this,bool *curB_flag,\
    float *P_0, float *P_out, float *ncross_dir, float s_len, int *flag, double *len_this,\
    float direction){
        unsigned long step_count = 0;
        //unsigned long step_lim = (MAX_STEP_RATIO*(BshapeN3.x+BshapeN3.y+BshapeN3.z));
        double len_lim = (MAX_STEP_RATIO*1.0*(BshapeN3.x+BshapeN3.y+BshapeN3.z));
        float p_mid, p1,p2; // for linear interpolation
        int flag_this;
        int dim_out;
        float3 PP1,PP2,B_P1,B_P2,B_Pstart, ncross_dir3,cur_P1;
        float3 P_tmp;
        double len_record=0;
        double twist=0;
        flag_this = 0;  // start from flag=0
        PP1=make_float3(P_0[0],P_0[1],P_0[2]);
        ncross_dir3=make_float3(ncross_dir[0],ncross_dir[1],ncross_dir[2]);
        B_Pstart = Interp3dxyzn(Bx,By,Bz,BshapeN3,PP1,true);
        

        while ( (flag_this==0) & (len_record<len_lim) & (step_count<MAX_NUM_STEP)){
            // trace Bline step by step
            P_tmp = RK4(Bx,By,Bz,BshapeN3,PP1, s_len*direction);
            PP2 = make_float3(P_tmp.x,P_tmp.y,P_tmp.z);
            len_record = len_record+lenVec3(PP1-PP2);
            if (curB_flag[0]){
                cur_P1 = Interp3dxyzn(curB_x,curB_y,curB_z,BshapeN3,PP1,false);
                B_P1 = Interp3dxyzn(Bx,By,Bz,BshapeN3,PP1,false);
                twist = twist+dot3(cur_P1,B_P1)/dot3(B_P1,B_P1)/4.0/M_PI*lenVec3(PP1-PP2);
            }

            flag_this = checkFlag(BshapeN3,PP2);  // check status
            if (flag_this>0){ // out of box
                len_record = len_record-lenVec3(PP1-PP2); // reverse step len
                if (curB_flag[0]){// reverse twist
                    twist = twist-dot3(cur_P1,B_P1)/dot3(B_P1,B_P1)/4.0/M_PI*lenVec3(PP1-PP2);
                }
                if (flag_this<=6){ // step out from surface
                    // linear estimation
                    dim_out = int((flag_this-1)/2);
                    p1 = selectFloat3xyz(PP1,dim_out);
                    p2 = selectFloat3xyz(PP2,dim_out);

                    if (fabsf(p1-p2)>1e-3){
                        if (flag_this%2==1){p_mid=0;} // step out from min surface
                        else{p_mid=float(selectInt3xyz(BshapeN3,dim_out));} // step out from max surface
                        B_P1 = Interp3dxyzn(Bx,By,Bz,BshapeN3,PP1,true);
                        B_P2 = Interp3dxyzn(Bx,By,Bz,BshapeN3,PP2,true);
                        if (fabsf(selectFloat3xyz(B_P1,dim_out))<0.2 | fabsf(selectFloat3xyz(B_P2,dim_out))<0.2){
                                P_out[0] = (PP1.x* (p2-p_mid) + PP2.x* (p_mid-p1))/(p2-p1);
                                P_out[1] = (PP1.y* (p2-p_mid) + PP2.y* (p_mid-p1))/(p2-p1);
                                P_out[2] = (PP1.z* (p2-p_mid) + PP2.z* (p_mid-p1))/(p2-p1); }
                        else{// rk4 to the surface
                            PP2 = RK4_boundary(Bx,By,Bz,BshapeN3,PP1,(p_mid-p1),dim_out);
                            P_out[0] = PP2.x;  P_out[1] = PP2.y;  P_out[2] = PP2.z;
                        }
                        len_record = len_record+fabsf(p_mid-p1)/(1e-4+fabsf(selectFloat3xyz(B_P1,dim_out)));    
                    }
                    else{
                        P_out[0] = PP1.x;  P_out[1] = PP1.y;  P_out[2] = PP1.z;
                    }

                    if (curB_flag[0]){
                        twist = twist+dot3(cur_P1,B_P1)/dot3(B_P1,B_P1)/4.0/M_PI \
                        *fabsf(p_mid-p1)/(1e-4+fabsf(selectFloat3xyz(B_P1,dim_out)));
                    }    
                }
                else{ // ignore
                    P_out[0] = PP2.x;  P_out[1] = PP2.y;  P_out[2] = PP2.z;
                }
            }
            PP1=PP2;
            step_count=step_count+1;
        }
        //printf("[%d][%f]:%f  :%f  :%f\n",step_count,P1[0],P1[1],P1[2]);
        len_this[0] = len_record;
        twist_this[0] = twist;
        flag[0] = flag_this;
    }




    __global__ void TraceAllBline(float *Bx,float *By,float *Bz,int *BshapeN,\
        float *curB_x, float *curB_y,  float *curB_z,double *twist,bool *curB_flag,\
        float *inp_x,float *inp_y, float *inp_z, float *inp_cross_dir,\
        float *start_x,float *start_y, float *start_z, int *flag_start,\
        float *end_x,  float *end_y,   float *end_z,   int *flag_end,\
        float *B_this_x,float *B_this_y, float *B_this_z, int *B_flag,\
        float *B_start_x,float *B_start_y, float *B_start_z,\
        float *B_end_x,float *B_end_y, float *B_end_z,\
        float *s_len,unsigned long long *N,double *LineLen){
            
            unsigned long long x = blockIdx.x * blockDim.x + threadIdx.x;
            unsigned long long y = blockIdx.y * blockDim.y + threadIdx.y; 
            unsigned long long idx_cur,dim_all,works_per_thread,Bline_ID,line_idx;
            int3 BshapeN3 = make_int3(BshapeN[0],BshapeN[1],BshapeN[2]);
    
            dim_all = (gridDim.x*blockDim.x*gridDim.y*blockDim.y); // upper lim 8192 
            idx_cur = (gridDim.x*blockDim.x) * y + x;                     
            works_per_thread = N[0]/dim_all+1;
            
            double *twist_this = new double[1];
            float *P_0 = new float[3];
            float *P_out = new float[3];
            int *flag_cur = new int[1];
            double *len_this = new double[1];
    
            for (line_idx=0; line_idx<works_per_thread; line_idx++){
                //Bline_ID = works_per_thread*idx_cur+line_idx;
                Bline_ID = idx_cur+line_idx*dim_all;
                if (Bline_ID<N[0]){
                    //printf("  %llu ",Bline_ID);
                    // forward
                    P_0[0] = inp_x[Bline_ID];
                    P_0[1] = inp_y[Bline_ID];
                    P_0[2] = inp_z[Bline_ID]; 
                    twist_this[0]=0;
                    TraceBline(Bx,By,Bz,BshapeN3,curB_x,curB_y,curB_z,twist_this,curB_flag,\
                        P_0, P_out,inp_cross_dir, s_len[0], flag_cur,len_this,1.0); // forward and backward
                    B_end_x[Bline_ID] = Interp3d(Bx,BshapeN3,P_out[0],P_out[1],P_out[2]);
                    B_end_y[Bline_ID] = Interp3d(By,BshapeN3,P_out[0],P_out[1],P_out[2]);
                    B_end_z[Bline_ID] = Interp3d(Bz,BshapeN3,P_out[0],P_out[1],P_out[2]);
                    end_x[Bline_ID] = P_out[0];
                    end_y[Bline_ID] = P_out[1];
                    end_z[Bline_ID] = P_out[2];
                    flag_end[Bline_ID] = flag_cur[0];
                    LineLen[Bline_ID] = len_this[0];
                    if (curB_flag[0]){twist[Bline_ID]=twist_this[0];}
                    //backward
                    P_0[0] = inp_x[Bline_ID];
                    P_0[1] = inp_y[Bline_ID];
                    P_0[2] = inp_z[Bline_ID]; 
                    twist_this[0]=0;
                    TraceBline(Bx,By,Bz,BshapeN3,curB_x,curB_y,curB_z,twist_this,curB_flag,\
                        P_0, P_out,inp_cross_dir, s_len[0], flag_cur,len_this,-1.0); // forward and backward
                    B_start_x[Bline_ID] = Interp3d(Bx,BshapeN3,P_out[0],P_out[1],P_out[2]);
                    B_start_y[Bline_ID] = Interp3d(By,BshapeN3,P_out[0],P_out[1],P_out[2]);
                    B_start_z[Bline_ID] = Interp3d(Bz,BshapeN3,P_out[0],P_out[1],P_out[2]);
                    start_x[Bline_ID] = P_out[0];
                    start_y[Bline_ID] = P_out[1];
                    start_z[Bline_ID] = P_out[2];
                    flag_start[Bline_ID] = flag_cur[0];
                    LineLen[Bline_ID] = LineLen[Bline_ID] + len_this[0];
                    if (curB_flag[0]){twist[Bline_ID]=twist[Bline_ID]+twist_this[0];}
                    //printf("[%d], %f, %f, %f\n", flag_out[idx_cur] ,out_x[idx_cur],out_y[idx_cur],out_z[idx_cur] );
                    
                    // record B in plane
                    B_this_x[Bline_ID] = Interp3d(Bx,BshapeN3,P_0[0],P_0[1],P_0[2]);
                    B_this_y[Bline_ID] = Interp3d(By,BshapeN3,P_0[0],P_0[1],P_0[2]);
                    B_this_z[Bline_ID] = Interp3d(Bz,BshapeN3,P_0[0],P_0[1],P_0[2]);
    
                    if (fabsf(B_this_x[Bline_ID]*inp_cross_dir[0]+B_this_y[Bline_ID]*inp_cross_dir[1]+B_this_z[Bline_ID]*inp_cross_dir[2])*100.\
                      <lenVec3xyz(B_this_x[Bline_ID],B_this_y[Bline_ID],B_this_z[Bline_ID])){
                        B_flag[Bline_ID] = 1;}
                    else{B_flag[Bline_ID] = 0;}
                    //printf("flag***:  %d  %d\n",flag_cur[0],flag_start[Bline_ID]);
                }
            }
            delete[] twist_this;
            delete[] P_0;
            delete[] P_out;
            delete[] flag_cur;
            delete[] len_this;
    }
    }