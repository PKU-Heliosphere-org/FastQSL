#include "hip/hip_runtime.h"
/**
*    @file TraceBline.cu
*    @author Peijin Zhang
*    The kernel of the Q-factor computation: tracing the magnetic field
*/
#include <math.h>
#include <stdio.h>
#include "hip/hip_vector_types.h"
#define M_PI 3.14159265    ///< Mathematical constant PI.
#define MAX_STEP_RATIO 16  ///< Maximum step length compared to box size.
#define TOL 1e-3 // toleranced error for each step [0.001~0.00001]

extern "C"{
#include "TraceBlineAdaptive.cuh"
inline __device__ float lenVec3xyz(float xx,float yy,float zz){
    return sqrtf(xx*xx + yy*yy + zz*zz);}

inline __device__ float lenVec3(float3 a){
    return sqrtf(a.x*a.x + a.y*a.y + a.z*a.z);}

inline __device__ float dot3(float3 a, float3 b)
{  return a.x*b.x + a.y*b.y + a.z*b.z;}

inline __device__ float3 divide3(float3 a, float b)
{  return make_float3(a.x/b , a.y/b , a.z/b);}

inline __device__ float3 normalize3(float4 a)
{  return make_float3(a.x/a.w,  a.y/a.w,  a.z/a.w);}

__forceinline__ __device__ float get_Idx3d(float *Arr,int3 AShapeN3,int Idx0,int Idx1,int Idx2){
    //return Arr[Idx0* AShapeN[1]*AShapeN[2]  +  Idx1* AShapeN[2]  +  Idx2];
    return Arr[Idx2* AShapeN3.y*AShapeN3.x  +  Idx1*AShapeN3.x  +  Idx0];
}

__global__ void  test_Idx3d(float *Arr,int *AShapeN, int *getIdx,float *res){
    printf("idx : %d    %d    %d   ", getIdx[0],getIdx[1],getIdx[2]);
    int3 AShapeN3 = make_int3(AShapeN[0],AShapeN[1],AShapeN[2]);
    res[0] = get_Idx3d(Arr,AShapeN3,getIdx[0],getIdx[1],getIdx[2]);
}

__device__ float Interp3d(float *Arr,int3 AShapeN3, \
    float inPoint_0, float inPoint_1, float inPoint_2){

    //algorithm [https://core.ac.uk/download/pdf/44386053.pdf]
    float rx,ry,rz; // ratio of the point
    float Arr000,Arr001,Arr010,Arr011,Arr100,Arr101,Arr110,Arr111;
    float Aget;
    int x_Idx,y_Idx,z_Idx;

    // handle out of boundary problem by extending
    inPoint_0 = (inPoint_0>0) ? inPoint_0 : 0.0001;
    inPoint_1 = (inPoint_1>0) ? inPoint_1 : 0.0001;
    inPoint_2 = (inPoint_2>0) ? inPoint_2 : 0.0001;
    inPoint_0 = (inPoint_0<AShapeN3.x-1) ? inPoint_0 : ((float)AShapeN3.x-1-0.0001);
    inPoint_1 = (inPoint_1<AShapeN3.y-1) ? inPoint_1 : ((float)AShapeN3.y-1-0.0001);
    inPoint_2 = (inPoint_2<AShapeN3.z-1) ? inPoint_2 : ((float)AShapeN3.z-1-0.0001);

    // ratio of the points to adjacent grid
    rx = inPoint_0-floorf(inPoint_0);
    ry = inPoint_1-floorf(inPoint_1);
    rz = inPoint_2-floorf(inPoint_2);

    // index of point in the down-side
    x_Idx = __float2int_rd(inPoint_0);
    y_Idx = __float2int_rd(inPoint_1);
    z_Idx = __float2int_rd(inPoint_2);
    
    //printf("x,y,z %f %f %f\n",inPoint_0,inPoint_1,inPoint_2);
    //printf("x,y,z %d %d %d\n",x_Idx,y_Idx,z_Idx);

    // grid boundary
    Arr000 = get_Idx3d(Arr,AShapeN3, x_Idx  ,y_Idx  ,z_Idx  );
    Arr001 = get_Idx3d(Arr,AShapeN3, x_Idx  ,y_Idx  ,z_Idx+1);
    Arr010 = get_Idx3d(Arr,AShapeN3, x_Idx  ,y_Idx+1,z_Idx  );
    Arr011 = get_Idx3d(Arr,AShapeN3, x_Idx  ,y_Idx+1,z_Idx+1);
    Arr100 = get_Idx3d(Arr,AShapeN3, x_Idx+1,y_Idx  ,z_Idx  );
    Arr101 = get_Idx3d(Arr,AShapeN3, x_Idx+1,y_Idx  ,z_Idx+1);
    Arr110 = get_Idx3d(Arr,AShapeN3, x_Idx+1,y_Idx+1,z_Idx  );
    Arr111 = get_Idx3d(Arr,AShapeN3, x_Idx+1,y_Idx+1,z_Idx+1);

    Aget =      Arr000* (1.-rx)*(1.-ry)*(1.-rz)+\
                Arr001* (1.-rx)*(1.-ry)*(  rz)+\
                Arr010* (1.-rx)*(  ry)*(1.-rz)+\
                Arr011* (1.-rx)*(  ry)*(  rz)+\
                Arr100* (  rx)*(1.-ry)*(1.-rz)+\
                Arr101* (  rx)*(1.-ry)*(  rz)+\
                Arr110* (  rx)*(  ry)*(1.-rz)+\
                Arr111* (  rx)*(  ry)*(  rz);
    return Aget;
}

inline __device__ float3 Interp3dxyzn(float *Arr_x,float *Arr_y,float *Arr_z,int3 AShapeN3, float3 inPoint_this){
    // normalized B interpolation
    //algorithm [https://core.ac.uk/download/pdf/44386053.pdf]
    float rx,ry,rz; // ratio of the point
    float Arr000_x,Arr001_x,Arr010_x,Arr011_x,Arr100_x,Arr101_x,Arr110_x,Arr111_x;
    float Arr000_y,Arr001_y,Arr010_y,Arr011_y,Arr100_y,Arr101_y,Arr110_y,Arr111_y;
    float Arr000_z,Arr001_z,Arr010_z,Arr011_z,Arr100_z,Arr101_z,Arr110_z,Arr111_z;
    float w000,  w001,  w010,  w011,  w100,  w101,  w110,  w111;
    int x_Idx,y_Idx,z_Idx;
    float norm_B;
    float3 res;

    float inPoint_0=inPoint_this.x;
    float inPoint_1=inPoint_this.y;
    float inPoint_2=inPoint_this.z;
    // ratio of the points to adjacent grid
    rx = inPoint_0-floorf(inPoint_0);
    ry = inPoint_1-floorf(inPoint_1);
    rz = inPoint_2-floorf(inPoint_2);
    // index of point in the down-side
    x_Idx = __float2int_rd(inPoint_0);
    y_Idx = __float2int_rd(inPoint_1);
    z_Idx = __float2int_rd(inPoint_2);
    
    if (inPoint_0<= 0.){
        rx=0; x_Idx=0;}
    if (inPoint_0>= AShapeN3.x -1){
        rx=1; x_Idx=AShapeN3.x -2;}
    if (inPoint_1<= 0.){
        ry=0; y_Idx=0;}
    if (inPoint_1>= AShapeN3.y -1){
        ry=1; y_Idx=AShapeN3.y -2;}
    if (inPoint_2<= 0.){
        rz=0.; z_Idx=0;}
    if (inPoint_2>= AShapeN3.z -1){
        rz=1.; z_Idx=AShapeN3.z -2;}
    
    // calculate the weight of the point and use for three times
    w000 = (1.-rx)*(1.-ry)*(1.-rz);
    w001 = (1.-rx)*(1.-ry)*(   rz);
    w010 = (1.-rx)*(   ry)*(1.-rz);
    w011 = (1.-rx)*(   ry)*(   rz);
    w100 = (   rx)*(1.-ry)*(1.-rz);
    w101 = (   rx)*(1.-ry)*(   rz);
    w110 = (   rx)*(   ry)*(1.-rz);
    w111 = (   rx)*(   ry)*(   rz);

    // grid boundary
    Arr000_x = get_Idx3d(Arr_x,AShapeN3, x_Idx  ,y_Idx  ,z_Idx  );
    Arr001_x = get_Idx3d(Arr_x,AShapeN3, x_Idx  ,y_Idx  ,z_Idx+1);
    Arr010_x = get_Idx3d(Arr_x,AShapeN3, x_Idx  ,y_Idx+1,z_Idx  );
    Arr011_x = get_Idx3d(Arr_x,AShapeN3, x_Idx  ,y_Idx+1,z_Idx+1);
    Arr100_x = get_Idx3d(Arr_x,AShapeN3, x_Idx+1,y_Idx  ,z_Idx  );
    Arr101_x = get_Idx3d(Arr_x,AShapeN3, x_Idx+1,y_Idx  ,z_Idx+1);
    Arr110_x = get_Idx3d(Arr_x,AShapeN3, x_Idx+1,y_Idx+1,z_Idx  );
    Arr111_x = get_Idx3d(Arr_x,AShapeN3, x_Idx+1,y_Idx+1,z_Idx+1);
    
    Arr000_y = get_Idx3d(Arr_y,AShapeN3, x_Idx  ,y_Idx  ,z_Idx  );
    Arr001_y = get_Idx3d(Arr_y,AShapeN3, x_Idx  ,y_Idx  ,z_Idx+1);
    Arr010_y = get_Idx3d(Arr_y,AShapeN3, x_Idx  ,y_Idx+1,z_Idx  );
    Arr011_y = get_Idx3d(Arr_y,AShapeN3, x_Idx  ,y_Idx+1,z_Idx+1);
    Arr100_y = get_Idx3d(Arr_y,AShapeN3, x_Idx+1,y_Idx  ,z_Idx  );
    Arr101_y = get_Idx3d(Arr_y,AShapeN3, x_Idx+1,y_Idx  ,z_Idx+1);
    Arr110_y = get_Idx3d(Arr_y,AShapeN3, x_Idx+1,y_Idx+1,z_Idx  );
    Arr111_y = get_Idx3d(Arr_y,AShapeN3, x_Idx+1,y_Idx+1,z_Idx+1);
    
    Arr000_z = get_Idx3d(Arr_z,AShapeN3, x_Idx  ,y_Idx  ,z_Idx  );
    Arr001_z = get_Idx3d(Arr_z,AShapeN3, x_Idx  ,y_Idx  ,z_Idx+1);
    Arr010_z = get_Idx3d(Arr_z,AShapeN3, x_Idx  ,y_Idx+1,z_Idx  );
    Arr011_z = get_Idx3d(Arr_z,AShapeN3, x_Idx  ,y_Idx+1,z_Idx+1);
    Arr100_z = get_Idx3d(Arr_z,AShapeN3, x_Idx+1,y_Idx  ,z_Idx  );
    Arr101_z = get_Idx3d(Arr_z,AShapeN3, x_Idx+1,y_Idx  ,z_Idx+1);
    Arr110_z = get_Idx3d(Arr_z,AShapeN3, x_Idx+1,y_Idx+1,z_Idx  );
    Arr111_z = get_Idx3d(Arr_z,AShapeN3, x_Idx+1,y_Idx+1,z_Idx+1);

    res.x=Arr000_x* w000+\
            Arr001_x* w001+\
            Arr010_x* w010+\
            Arr011_x* w011+\
            Arr100_x* w100+\
            Arr101_x* w101+\
            Arr110_x* w110+\
            Arr111_x* w111;
    res.y=Arr000_y* w000+\
            Arr001_y* w001+\
            Arr010_y* w010+\
            Arr011_y* w011+\
            Arr100_y* w100+\
            Arr101_y* w101+\
            Arr110_y* w110+\
            Arr111_y* w111;
    res.z=Arr000_z* w000+\
            Arr001_z* w001+\
            Arr010_z* w010+\
            Arr011_z* w011+\
            Arr100_z* w100+\
            Arr101_z* w101+\
            Arr110_z* w110+\
            Arr111_z* w111;
    norm_B = sqrtf(res.x*res.x+res.y*res.y+res.z*res.z);
    res.x = res.x/norm_B;
    res.y = res.y/norm_B;
    res.z = res.z/norm_B;
    return res;
}


__global__ void test_Interp3dxyz(float *Arr_x,float *Arr_y,float *Arr_z,int *AShapeN, \
    float *inPoint_0, float *inPoint_1, float *inPoint_2){
        float3 res,pointthis;
        pointthis=make_float3(inPoint_0[0],inPoint_1[0],inPoint_2[0]);
        int3 shapeshape = make_int3(AShapeN[0],AShapeN[1],AShapeN[2]);
        res = Interp3dxyzn(Arr_x,Arr_y,Arr_z,shapeshape,pointthis);
    }

/**
* Adaptive step-size integral scheme
*/
inline __device__ float4 RKF45(float *Bx,float *By,float *Bz,int3 BshapeN3, float3 P0, float s_len){
    float3 k1,k2,k3,k4,k5,k6;
    float3 P_a,P_b;
    float4 res_end;
    float err_step; 

    // parameters of the Butcher tableau
    //float c2,c3,c4,c5,c6;
    // c need  to be included only when f' depends on x
    float a21,a31,a32,a41,a42,a43,a51,a52,a53,a54,a61,a62,a63,a64,a65;
    float b1,b2,b3,b4,b5,b6;
    float bb1,bb2,bb3,bb4,bb5; 
    float ce1,ce3,ce4,ce5,ce6;
    //c2=1./4.;   c3 = 3./8.;  c4=12./13.; c5=1.;  c6=1./2.; 
    a21=1./4.;
    a31=3./32.;      a32=9./32.;
    a41=1932./2197.; a42=-7200./2197.; a43= 7296./2197.;
    a51=439./216.;   a52= -8.;          a53=  3680./513.;   a54=-845./4104.;
    a61= -8./27.;    a62= 2.;           a63= -3544./2565.;  a64= 1859./4104.; a65= -11./40.;
    b1 = 16./135.;   b2 =0.;  b3 = 6656./12825.;   b4 = 28561./56430.; b5 = -9./50.;  b6 = 2./55.; 
    bb1= 25./216.;   bb2=0.;  bb3 = 1408./2565.; bb4 = 2197./4104.;  bb5 = -1./5.;
    ce1 = 1./360.;     ce3 = -128./4275.;    ce4 = -2197./75240.;   ce5 = 1./50.;   ce6 = 2./55.;
    k1 = s_len*Interp3dxyzn(Bx,By,Bz,BshapeN3,P0);
    k2 = s_len*Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+ (a21*k1));
    k3 = s_len*Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+ (a31*k1+ a32*k2));
    k4 = s_len*Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+ (a41*k1+ a42*k2+ a43*k3));
    k5 = s_len*Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+ (a51*k1+ a52*k2+ a53*k3+ a54*k4));
    k6 = s_len*Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+ (a61*k1+ a62*k2+ a63*k3+ a64*k4+ a65*k5));
    
    P_a = P0+ (b1*k1  +b2*k2  +b3*k3+  b4*k4  +b5*k5  +b6*k6);
    //P_b = P0+ (bb1*k1 +bb2*k2 +bb3*k3+ bb4*k4 +bb5*k5);

    //err_step = lenVec3(P_a-P_b);
    err_step = lenVec3(ce1*k1+ce3*k3+ce4*k4+ce5*k5+ce6*k6);
    res_end.x = P_a.x;
    res_end.y = P_a.y;
    res_end.z = P_a.z;
    res_end.w = err_step;
    return res_end;
}

/**
* 4-stage Runge-Kutta integral function
*/
inline __device__ float3 RK4(float *Bx,float *By,float *Bz,int3 BshapeN3, float3 P0, float s_len){
    float3 k1,k2,k3,k4,P_end;
    k1 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0);
    k2 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k1/2.);
    k3 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k2/2.);
    k4 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k3);
    P_end = P0+1./6.*s_len*(k1 +2.*k2 +2.*k3 +k4);
    return P_end;
}

inline __device__ float selectFloat3xyz(float3 a, int dim){
    float res;
    switch(dim) {
        case 0  : res = a.x;  break;
        case 1  : res = a.y;  break;
        case 2  : res = a.z;  break;}
    return res;
}
inline __device__ float selectInt3xyz(int3 a, int dim){
    int res;
    switch(dim) {
        case 0  : res = a.x;  break;
        case 1  : res = a.y;  break;
        case 2  : res = a.z;  break;}
    return res;
}

inline __device__ float3 RK4_boundary(float *Bx,float *By,float *Bz,int3 BshapeN3, float3 P0, float s_len,int b_dim){
    float B0_k1,B0_k2,B0_k3,B0_k4;
    float3 Bk1,Bk2,Bk3,Bk4,P_end,k1,k2,k3,k4;
    // k1
    Bk1 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0);
    B0_k1 = selectFloat3xyz(Bk1,b_dim);
    k1 = Bk1/B0_k1;
    // k2
    Bk2 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k1/2.);
    B0_k2 = selectFloat3xyz(Bk2,b_dim);
    k2 = Bk2/B0_k2;
    // k3
    Bk3  = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k2/2.);
    B0_k3 = selectFloat3xyz(Bk3,b_dim);
    k3 = Bk3/B0_k3;
    // k4
    Bk4 = Interp3dxyzn(Bx,By,Bz,BshapeN3,P0+s_len*k3);
    B0_k4 = selectFloat3xyz(Bk4,b_dim);
    k4 = Bk4/B0_k4;
    P_end = P0 + (1./6.)* s_len*( k1 + 2.0*k2 + 2.0*k3 + k4);
    return P_end;
}

inline __device__ int checkFlag(int3 BshapeN3, float3 P_cur){
    // check current status
    int flag_res = 42; // 42 means un-categorized
    // flag=0 means inside running box
    if (P_cur.x>=0. &P_cur.y>0. &P_cur.z>=0. &  \
        P_cur.x<=BshapeN3.x-1. &P_cur.y<=BshapeN3.y-1. & P_cur.z<=BshapeN3.z-1. ){flag_res=0;} 
    else{ // ouside
        if (P_cur.x< 0.             &P_cur.y>=0.            &P_cur.z>=0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=1;} // x min 
        if (P_cur.x>=0.             &P_cur.y>=0.            &P_cur.z>=0. &  \
            P_cur.x>=BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=2;} // x max
        if (P_cur.x>=0.             &P_cur.y< 0.            &P_cur.z>=0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=3;} // y min
        if (P_cur.x>=0.             &P_cur.y>=0.            &P_cur.z>=0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y>=BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=4;} // y max 
        if (P_cur.x>=0.             &P_cur.y>=0.            &P_cur.z< 0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z< BshapeN3.z-1.   ){flag_res=5;} // z min
        if (P_cur.x>=0.             &P_cur.y>=0.            &P_cur.z>=0. &  \
            P_cur.x< BshapeN3.x-1.  &P_cur.y< BshapeN3.y-1. &P_cur.z>=BshapeN3.z-1.   ){flag_res=6;} // z max
    }
    // dim = int((flag-1)/2)
    return flag_res;
}

__global__ void testTraceBlineAdap(float *Bx,float *By,float *Bz,int* BshapeN3,\
    float *P_0, float *P_out, float *ncross_dir, float* s_len, int *flag, double *len_this){
        float tol_coef = 1.0;
        TraceBlineAdap(Bx,By,Bz, make_int3(BshapeN3[0],BshapeN3[1],BshapeN3[2]),\
         P_0, P_out,ncross_dir, s_len[0], flag, len_this,1.0,tol_coef);
    }
__device__ void TraceBlineAdap(float *Bx,float *By,float *Bz,int3 BshapeN3,\
    float *P_0, float *P_out, float *ncross_dir, float s_len, int *flag, double *len_this,\
    float direction,float tol_coef){
        unsigned long step_count = 0;
        unsigned long step_lim = (MAX_STEP_RATIO*2*(BshapeN3.x+BshapeN3.y+BshapeN3.z));
        float scale,tol_this;
        float p_mid, p1,p2; // for linear interpolation
        int flag_this;
        int dim_out;
        float3 PP1,PP2,B_P1,B_P2,B_Pstart, ncross_dir3;
        float4 P_tmp;
        double len_record=0;
        flag_this = 0;  // start from flag=0
        PP1=make_float3(P_0[0],P_0[1],P_0[2]);
        ncross_dir3=make_float3(ncross_dir[0],ncross_dir[1],ncross_dir[2]);
        B_Pstart = Interp3dxyzn(Bx,By,Bz,BshapeN3,PP1);

        if (fabsf(dot3(B_Pstart,ncross_dir3))<=0.05){tol_this=TOL/8e3;}
        else {tol_this=TOL*powf(fabsf(dot3(B_Pstart,ncross_dir3)),3);}
        
        tol_this=tol_this*tol_coef;
        while ( (flag_this==0) & (step_count<step_lim)){
            // trace Bline step by step
            P_tmp = RKF45(Bx,By,Bz,BshapeN3,PP1, s_len*direction);
            PP2 = make_float3(P_tmp.x,P_tmp.y,P_tmp.z);
            scale = powf(tol_this/P_tmp.w/2.,0.25);
            if (scale<0.618){s_len = s_len*0.618;// redo RK45 when the error is too large
                continue; }
            s_len = s_len*scale;
            if (s_len>100.)  {s_len=100.;} // upper limit of the step size
            if (s_len<1./10.){s_len=1./10.;} //lower limit of the step size
            //len_record = len_record+s_len;
            len_record = len_record+lenVec3(PP1-PP2);
            flag_this = checkFlag(BshapeN3,PP2);  // check status
            if (flag_this>0){ // out of box
                len_record = len_record-lenVec3(PP1-PP2); // reverse step len
                if (flag_this<=6){ // step out from surface
                    // linear estimation
                    dim_out = int((flag_this-1)/2);
                    p1 = selectFloat3xyz(PP1,dim_out);
                    p2 = selectFloat3xyz(PP2,dim_out);
                    if (flag_this%2==1){p_mid=0;} // step out from min surface
                    else{p_mid=float(selectInt3xyz(BshapeN3,dim_out));} // step out from max surface
                    B_P1 = Interp3dxyzn(Bx,By,Bz,BshapeN3,PP1);
                    B_P2 = Interp3dxyzn(Bx,By,Bz,BshapeN3,PP2);
                    if (fabsf(selectFloat3xyz(B_P1,dim_out))<0.2 | fabsf(selectFloat3xyz(B_P2,dim_out))<0.2){
                            P_out[0] = (PP1.x* (p2-p_mid) + PP2.x* (p_mid-p1))/(p2-p1);
                            P_out[1] = (PP1.y* (p2-p_mid) + PP2.y* (p_mid-p1))/(p2-p1);
                            P_out[2] = (PP1.z* (p2-p_mid) + PP2.z* (p_mid-p1))/(p2-p1); }
                    else{// rk4 to the surface
                        PP2 = RK4_boundary(Bx,By,Bz,BshapeN3,PP1,(p_mid-p1),dim_out);
                        P_out[0] = PP2.x;  P_out[1] = PP2.y;  P_out[2] = PP2.z;
                    }
                    len_record = len_record+fabsf(p_mid-p1)/(1e-4+fabsf(selectFloat3xyz(B_P1,dim_out)));
                }
                else{ // ignore
                    P_out[0] = PP2.x;  P_out[1] = PP2.y;  P_out[2] = PP2.z;
                }
            }
            PP1=PP2;
            step_count=step_count+1;
        
        
        }
        //printf("[%d][%f]:%f  :%f  :%f\n",step_count,P1[0],P1[1],P1[2]);
        len_this[0] = len_record;
        flag[0] = flag_this;
    }


__global__ void test_Interp3d(float *Arr,int *AShapeN, float *inPoint,float *res){
    int3 AShapeN3 = make_int3(AShapeN[0],AShapeN[1],AShapeN[2]);
    res[0] = Interp3d(Arr,AShapeN3,inPoint[0],inPoint[1],inPoint[2]);
}

__global__ void TraceAllBline(float *Bx,float *By,float *Bz,int *BshapeN,\
    float *inp_x,float *inp_y, float *inp_z, float *inp_cross_dir,\
    float *start_x,float *start_y, float *start_z, int *flag_start,\
    float *end_x,  float *end_y,   float *end_z,   int *flag_end,\
    float *B_this_x,float *B_this_y, float *B_this_z, int *B_flag,\
    float *B_start_x,float *B_start_y, float *B_start_z,\
    float *B_end_x,float *B_end_y, float *B_end_z,\
    float *s_len,unsigned long long *N,double *LineLen,float *tol_coef){
        
        unsigned long long x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned long long y = blockIdx.y * blockDim.y + threadIdx.y; 
        unsigned long long idx_cur,dim_all,works_per_thread,Bline_ID,line_idx;
        int3 BshapeN3 = make_int3(BshapeN[0],BshapeN[1],BshapeN[2]);

        dim_all = (gridDim.x*blockDim.x*gridDim.y*blockDim.y); // upper lim 8192 
        idx_cur = (gridDim.x*blockDim.x) * y + x;                     
        works_per_thread = N[0]/dim_all+1;
        
        float *P_0 = new float[3];
        float *P_out = new float[3];
        int *flag_cur = new int[1];
        double *len_this = new double[1];

        for (line_idx=0; line_idx<works_per_thread; line_idx++){
            //Bline_ID = works_per_thread*idx_cur+line_idx;
            Bline_ID = idx_cur+line_idx*dim_all;
            if (Bline_ID<N[0]){
                //printf("  %llu ",Bline_ID);
                // forward
                P_0[0] = inp_x[Bline_ID];
                P_0[1] = inp_y[Bline_ID];
                P_0[2] = inp_z[Bline_ID]; 
                TraceBlineAdap(Bx,By,Bz,BshapeN3,P_0, P_out,inp_cross_dir, s_len[0], flag_cur,len_this,1.0,tol_coef[0]); // forward and backward
                B_end_x[Bline_ID] = Interp3d(Bx,BshapeN3,P_out[0],P_out[1],P_out[2]);
                B_end_y[Bline_ID] = Interp3d(By,BshapeN3,P_out[0],P_out[1],P_out[2]);
                B_end_z[Bline_ID] = Interp3d(Bz,BshapeN3,P_out[0],P_out[1],P_out[2]);
                end_x[Bline_ID] = P_out[0];
                end_y[Bline_ID] = P_out[1];
                end_z[Bline_ID] = P_out[2];
                flag_end[Bline_ID] = flag_cur[0];
                LineLen[Bline_ID] = len_this[0];
                //backward
                P_0[0] = inp_x[Bline_ID];
                P_0[1] = inp_y[Bline_ID];
                P_0[2] = inp_z[Bline_ID]; 
                TraceBlineAdap(Bx,By,Bz,BshapeN3,P_0, P_out,inp_cross_dir, s_len[0], flag_cur,len_this,-1.0,tol_coef[0]); // forward and backward
                B_start_x[Bline_ID] = Interp3d(Bx,BshapeN3,P_out[0],P_out[1],P_out[2]);
                B_start_y[Bline_ID] = Interp3d(By,BshapeN3,P_out[0],P_out[1],P_out[2]);
                B_start_z[Bline_ID] = Interp3d(Bz,BshapeN3,P_out[0],P_out[1],P_out[2]);
                start_x[Bline_ID] = P_out[0];
                start_y[Bline_ID] = P_out[1];
                start_z[Bline_ID] = P_out[2];
                flag_start[Bline_ID] = flag_cur[0];
                LineLen[Bline_ID] = LineLen[Bline_ID] + len_this[0];
                //printf("[%d], %f, %f, %f\n", flag_out[idx_cur] ,out_x[idx_cur],out_y[idx_cur],out_z[idx_cur] );
                
                // record B in plane
                B_this_x[Bline_ID] = Interp3d(Bx,BshapeN3,P_0[0],P_0[1],P_0[2]);
                B_this_y[Bline_ID] = Interp3d(By,BshapeN3,P_0[0],P_0[1],P_0[2]);
                B_this_z[Bline_ID] = Interp3d(Bz,BshapeN3,P_0[0],P_0[1],P_0[2]);
                
                if (fabsf(B_this_x[Bline_ID]*inp_cross_dir[0]+B_this_y[Bline_ID]*inp_cross_dir[1]+B_this_z[Bline_ID]*inp_cross_dir[2])*100.\
                  <lenVec3xyz(B_this_x[Bline_ID],B_this_y[Bline_ID],B_this_z[Bline_ID])){
                    B_flag[Bline_ID] = 1;}
                else{B_flag[Bline_ID] = 0;}
                //printf("flag***:  %d  %d\n",flag_cur[0],flag_start[Bline_ID]);
            }
        }
        
        delete[] P_0;
        delete[] P_out;
        delete[] flag_cur;
        delete[] len_this;
}
}