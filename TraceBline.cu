
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#ifndef M_PI
#define M_PI 3.14159265
#endif

__device__ float lenVec3(float xx,float yy,float zz){return sqrt(xx*xx+yy*yy+zz*zz);}

__device__ float get_Idx3d(float *Arr,int *AShapeN,int xIdx,int yIdx,int zIdx){
    return Arr[xIdx* AShapeN[1]*AShapeN[2]  +  yIdx* AShapeN[2]  +  zIdx];}

__device__ float Interp3d(float *Arr,int *AShapeN, \
    float inPoint_x, float inPoint_y, float inPoint_z){

    //algorithm [https://core.ac.uk/download/pdf/44386053.pdf]
    float rx,ry,rz; // ratio of the point
    float Arr000,Arr001,Arr010,Arr011,Arr100,Arr101,Arr110,Arr111;
    float Aget;
    int x_Idx,y_Idx,z_Idx;
    
    // handle out of boundary problem by extending
    inPoint_x = (inPoint_x>0) ? inPoint_x : 0;
    inPoint_y = (inPoint_y>0) ? inPoint_y : 0;
    inPoint_z = (inPoint_z>0) ? inPoint_z : 0;
    inPoint_x = (inPoint_x<AShapeN[0]) ? inPoint_x : AShapeN[0];
    inPoint_y = (inPoint_y<AShapeN[1]) ? inPoint_y : AShapeN[1];
    inPoint_z = (inPoint_z<AShapeN[2]) ? inPoint_z : AShapeN[2];

    // ratio of the points to adjacent grid
    rx = inPoint_x-floorf(inPoint_x);
    ry = inPoint_y-floorf(inPoint_y);
    rz = inPoint_z-floorf(inPoint_z);

    // index of point in the down-side
    x_Idx = __float2int_rd(inPoint_x);
    y_Idx = __float2int_rd(inPoint_y);
    z_Idx = __float2int_rd(inPoint_z);

    // grid boundary
    Arr000 = get_Idx3d(Arr,AShapeN, x_Idx  ,y_Idx  ,z_Idx  );
    Arr001 = get_Idx3d(Arr,AShapeN, x_Idx  ,y_Idx  ,z_Idx+1);
    Arr010 = get_Idx3d(Arr,AShapeN, x_Idx  ,y_Idx+1,z_Idx  );
    Arr011 = get_Idx3d(Arr,AShapeN, x_Idx  ,y_Idx+1,z_Idx+1);
    Arr100 = get_Idx3d(Arr,AShapeN, x_Idx+1,y_Idx  ,z_Idx  );
    Arr101 = get_Idx3d(Arr,AShapeN, x_Idx+1,y_Idx  ,z_Idx+1);
    Arr110 = get_Idx3d(Arr,AShapeN, x_Idx+1,y_Idx+1,z_Idx  );
    Arr111 = get_Idx3d(Arr,AShapeN, x_Idx+1,y_Idx+1,z_Idx+1);

    Aget =      Arr000* (1.-rx)*(1.-ry)*(1.-rz)+\
                Arr001* (1.-rx)*(1.-ry)*(  rz)+\
                Arr010* (1.-rx)*(  ry)*(1.-rz)+\
                Arr011* (1.-rx)*(  ry)*(  rz)+\
                Arr100* (  rx)*(1.-ry)*(1.-rz)+\
                Arr101* (  rx)*(1.-ry)*(  rz)+\
                Arr110* (  rx)*(  ry)*(1.-rz)+\
                Arr111* (  rx)*(  ry)*(  rz);
    
    return Aget;
}

__device__ void stepForward(float *Bx,float *By,float *Bz,int *BshapeN,\
        float *P_start, float *P_end, float s_len, int *flag){
    float Bx_cur,By_cur,Bz_cur,B0;
    Bx_cur  = Interp3d(Bx,BshapeN,P_start[0],P_start[1],P_start[2]);
    By_cur  = Interp3d(Bx,BshapeN,P_start[0],P_start[1],P_start[2]);
    Bz_cur  = Interp3d(Bx,BshapeN,P_start[0],P_start[1],P_start[2]);
    B0 = lenVec3(Bx_cur,By_cur,Bz_cur);
    P_end[0] = P_start[0]+s_len*Bx_cur/B0;
    P_end[1] = P_start[1]+s_len*By_cur/B0;
    P_end[2] = P_start[2]+s_len*Bz_cur/B0;
}

__device__ void RK4(float *Bx,float *By,float *Bz,float *BshapeN,\
    float *P_start, float *P_end, float s_len, int *flag){}

__device__ int checkFlag(float *BshapeN, float *P_cur){
    // check current status
    int flag_res = 42; // 42 means un-categorized
    // flag=0 means inside running box
    if (P_cur[0]>0 &P_cur[1]>0 &P_cur[2]>0 &  \
        P_cur[0]<BshapeN[0] &P_cur[1]<BshapeN[1]&P_cur[2]<BshapeN[2] ){flag_res=0;} 
    // flag=1 means outside box below (normal end of simulation)
    if (P_cur[0]>0 &P_cur[1]>0 &P_cur[2]<0 &  \
        P_cur[0]<BshapeN[0] &P_cur[1]<BshapeN[1]&P_cur[2]<BshapeN[2] ){flag_res=1;} 

    return flag_res;
}

__global__ void test_Idx3d(float *Arr,int *AShapeN, int *getIdx,float *res){
    res[0] = get_Idx3d(Arr,AShapeN,getIdx[0],getIdx[1],getIdx[2]);
}

__global__ void test_Interp3d(float *Arr,int *AShapeN, float *inPoint,float *res){
    res[0] = Interp3d(Arr,AShapeN,inPoint[0],inPoint[1],inPoint[2]);
}



__global__ void TraceBline(float *Bx,float *By,float *Bz,\
    float *inp_x,float *inp_y, float *inp_z,\
    float *out_x,float *out_y, float *out_z, \
    int *flag_out,unsigned long long int N){
        
        float x_cur,y_cur,z_cur;

        unsigned long long x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned long long y = blockIdx.y * blockDim.y + threadIdx.y; 
        unsigned long long idx_cur = (gridDim.x*blockDim.x) * y + x;     
        if (idx_cur<N){
            x_cur = inp_x[idx_cur];
            y_cur = inp_y[idx_cur];
            z_cur = inp_z[idx_cur];
            // main procedure of B-line tracking 
        }
}
